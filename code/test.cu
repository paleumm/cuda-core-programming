
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void add_vector(int *a, int *b, int *c)
{
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
    return;
}

int main()
{
    int a[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int b[] = {4, 5, 6, 7, 8, 9, 10, 11, 12, 13};

    int c[sizeof(a) / sizeof(int)] = {0};

    int *cuda_a = 0;
    int *cuda_b = 0;
    int *cuda_c = 0;

    // allocate memory in GPU
    hipMalloc(&cuda_a, sizeof(a));
    hipMalloc(&cuda_b, sizeof(b));
    hipMalloc(&cuda_c, sizeof(c));

    // copy vector to GPU's memory
    hipMemcpy(cuda_a, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(b), hipMemcpyHostToDevice);

    // add_vector <<< GRID_SIZE, BLOCK_SIZE >>> (parameters);
    add_vector<<<1, sizeof(a) / sizeof(int)>>>(cuda_a, cuda_b, cuda_c);

    hipMemcpy(c, cuda_c, sizeof(c), hipMemcpyDeviceToHost);

    return;
}