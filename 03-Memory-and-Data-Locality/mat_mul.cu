/*
 * =====================================================================================
 *
 *       Filename:  mat_mul.cu
 *    Description:  matrix multiplication
 *
 *        Version:  1.0
 *        Created:  16/07/2022 19:13:52
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Permpoon B (pb)
 *        Company:  none
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32

// Low performance Matrix Multiplication
__global__ void mat_mul_kernel(float *M, float *N, float *P, int width){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < width) && (col < width)){
        float Pval = 0;

        // each iteration performed 2 global mem access and 1 add and mul.
        for (int k = 0; k < width; k++)
            Pval += M[row * width + k] * N[k * width + col];

        // write the Pval to index if output matrix
        P[row * width + col] = Pval;
    }
}

__global__ void mat_mul_tiles(float *d_M, float *d_N, float *d_P, int width){

    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float Pval = 0;

    // Strip-mining => break into phases
    for(int tile = 0; tile < ceil(width/(float)TILE_WIDTH); tile++){
        // Assign the tiles for each thread. 
        // d_M(y,x) => Mds(y,x), d_N(y,x) => Nds(y,x)
        if((row < width) && (tile * TILE_WIDTH + tx)<width)
            Mds[ty][tx] = d_M[row * width + tile * TILE_WIDTH + tx];
        if((tile*TILE_WIDTH + ty) < width && (col < width))
            Nds[ty][tx] = d_N[(tile * TILE_WIDTH +ty)*width + col];

        __syncthreads(); // wait for all thread to load tile

        // Mds(y,k)*Nds(k,x) => Pval
        for(int k = 0; k < TILE_WIDTH; k++){
            Pval += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads(); // ensure that all thread finished using tile
    }

    if((row < width) && (col < width)) d_P[row*width + col] = Pval;
}

__host__ void mul_mat(float *h_out, float *h_mat1, float *h_mat2, int n){
    // size of matrix
    int mat_sz = n * n * sizeof(float);
    float *d_first, *d_second, *d_out;

    hipError_t errFirst = hipMalloc((void**)&d_first, mat_sz);
    if(errFirst != hipSuccess){
       printf("%s in %s at line %d\n",hipGetErrorString(errFirst), __FILE__, __LINE__);
       exit(EXIT_FAILURE);
    }
    hipMemcpy(d_first, h_mat1, mat_sz, hipMemcpyHostToDevice);

    hipError_t errSecond = hipMalloc((void**)&d_second, mat_sz);
    if(errSecond != hipSuccess){
       printf("%s in %s at line %d\n",hipGetErrorString(errSecond), __FILE__, __LINE__);
       exit(EXIT_FAILURE);
    }
    hipMemcpy(d_second, h_mat2, mat_sz, hipMemcpyHostToDevice);

    hipError_t errOut = hipMalloc((void**)&d_out, mat_sz);  
    if(errOut != hipSuccess){
       printf("%s in %s at line %d\n",hipGetErrorString(errOut), __FILE__, __LINE__);
       exit(EXIT_FAILURE);
    }

    dim3 dimBlock(TILE_WIDTH , TILE_WIDTH);
    dim3 dimGrid(ceil(n/float(dimBlock.x)), ceil(n/float(dimBlock.y)));
    mat_mul_tiles<<<dimGrid, dimBlock>>>(d_first, d_second, d_out, n);
    hipMemcpy(h_out, d_out, mat_sz, hipMemcpyDeviceToHost);

    hipFree(d_first); hipFree(d_second); hipFree(d_out);

}



int main(int argc, char* argv[]){
    // code section
    // executable parameter
    int n = strtol(argv[1], NULL, 10);

    int size = n*n;
    
    // initialize matrix
    float A[size], B[size], C[size];
    
    for (size_t i = 0; i < size; i++){
        B[i] = (i%n)+1;
        C[i] = (i%n)+1;
        //B[i] = 1;
        //C[i] = 1;
        //if(i%n==0) printf("\n");
        //printf("%.0f ",B[i]);
    }
    printf("\n");
    
    for(int i = 0 ; i < 100; i++){
        mul_mat(A, B, C, n);
    }
    //mul_mat(A, B, C, n);

    // printf("Output = ");
    // for( int i = 0; i < size ;i++){
    //     if(i%n==0) printf("\n");
    //     printf("%.0f ",A[i]);
    // }
    // printf("\n");

    return 0;
}